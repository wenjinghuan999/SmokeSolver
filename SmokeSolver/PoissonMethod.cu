#include "hip/hip_runtime.h"

#include "common.cuh"
#include "PoissonMethod.h"
using namespace ssv;

namespace
{
	using ssv::uint;

	// Red-black Gauss-Seidel
	// LAUNCH : block (ny - 2), thread (nx - 2)
	// p : nx x ny
	// g : nx x ny
	// omega : SOR coefficient
	// redblack : 0 or 1 indicating red or black
	template <typename QType>
	__global__ void kernelGS2d(
		BlobWrapper<QType> q, BlobWrapperConst<QType> g,
		T omega, uint redblack
	)
	{
		uint y = blockIdx.x * 2u + 1u;
		uint x = threadIdx.x + 1u;

		// Red - all cells with (x + y) % 2 == 0
		y += (x & 1) ^ redblack;

		QType v = q(x - 1u, y);
		v += q(x + 1u, y);
		v += q(x, y - 1u);
		v += q(x, y + 1u);

		q(x, y) = omega * (v - g(x, y)) / (T)(4) + ((T)(1) - omega) * q(x, y);
	}

	// Red-black Gauss-Seidel
	// LAUNCH : block (ny - 2, (nz - 2) / 2), thread (nx - 2)
	// p : nx x ny x nz
	// g : nx x ny x nz
	// omega : SOR coefficient
	// redblack : 0 or 1 indicating red or black
	template <typename QType>
	__global__ void kernelGS3d(
		BlobWrapper<QType> q, BlobWrapperConst<QType> g,
		T omega, uint redblack
	)
	{
		uint z = blockIdx.y * 2u + 1u;
		uint y = blockIdx.x + 1u;
		uint x = threadIdx.x + 1u;

		// Red - all cells with (x + y + z) % 2 == 0
		z += ((x + y) & 1) ^ redblack;

		QType v = q(x - 1u, y, z);
		v += q(x + 1u, y, z);
		v += q(x, y - 1u, z);
		v += q(x, y + 1u, z);
		v += q(x, y, z - 1u);
		v += q(x, y, z + 1u);

		q(x, y, z) = omega * (v - g(x, y, z)) / (T)(6) 
			+ ((T)(1) - omega) * q(x, y, z);
	}
}

template<typename QType>
void PoissonMethodGS<QType>::operator()(
	Blob<QType>& q, const Blob<QType>& g
	) const
{
	if (q.nz() < 3u)
	{
		for (uint i = 0; i < _iterations; i++)
		{
			kernelGS2d<<<(q.ny() - 2u) / 2u, q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 0
			);
			kernelGS2d<<<(q.ny() - 2u) / 2u, q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 1u
			);
		}
	}
	else
	{
		for (uint i = 0; i < _iterations; i++)
		{
			kernelGS3d<<<dim3(q.ny() - 2u, (q.nz() - 2u) / 2u), q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 0
			);
			kernelGS3d<<<dim3(q.ny() - 2u, (q.nz() - 2u) / 2u), q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 1u
			);
		}
	}
}

template class PoissonMethodGS<T>;
template class PoissonMethodGS<T2>;
template class PoissonMethodGS<T4>;

namespace
{
	using ssv::uint;

	// Down sampling
	// LAUNCH : block (ny), thread (nx)
	// qout :  nx   x  ny   (with border)
	// qin  : 2nx-2 x 2ny-2 (with border)
	template <typename QType>
	__global__ void kernelDownSample2d(
		BlobWrapper<QType> qout, BlobWrapperConst<QType> qin
	)
	{
		uint y = blockIdx.x;
		uint x = threadIdx.x;

	}
}

template <typename QType>
Blob<QType>::shape_t PoissonMethodVCycle<QType>::_NextShape(
	const Blob<QType>::shape_t &shape
)
{
	uint nx, ny, nz;
	std::tie(nx, ny, nz) = shape;
	nx = (nx + 1u) / 2u + 1u; if (nx < 3u) nx = 3u;
	ny = (ny + 1u) / 2u + 1u; if (ny < 3u) ny = 3u;
	if (nz >= 3u)
	{
		nz = (nz + 1u) / 2u + 1u; if (nz < 3u) nz = 3u;
	}
	return std::make_tuple(nx, ny, nz);
}

template <typename QType>
void PoissonMethodVCycle<QType>::_DownSample(
	Blob<QType> &qout, const Blob<QType> &qin
)
{
	assert(qout.shape() == _NextShape(qin.shape()));
	
}

template <typename QType>
void PoissonMethodVCycle<QType>::_UpSample(
	Blob<QType> &qout, const Blob<QType> &qin
)
{
	assert(qin.shape() == _NextShape(qout.shape()));

}

template<typename QType>
void PoissonMethodVCycle<QType>::operator()(Blob<QType>& q, const Blob<QType>& g) const
{
	typename Blob<QType>::shape_t shape = q.shape();
	shape = _NextShape(shape);

	if (_buffers.empty() || _buffers.front().shape() != shape)
	{
		_buffers.clear();
		for (uint i = 0; i < _levels; i++)
		{
			_buffers.emplace_back(shape, q.gpu_device(), false);
			shape = _NextShape(shape);
		}

		shape = _NextShape(q.shape());
	}


}

template class PoissonMethodVCycle<T>;
