#include "hip/hip_runtime.h"

#include "common.cuh"
#include "PoissonMethod.h"
using namespace ssv;

namespace
{
	using ssv::uint;

	// Red-black Gauss-Seidel
	// LAUNCH : block (ny - 2), thread (nx - 2)
	// p : nx x ny
	// g : nx x ny
	// omega : SOR coefficient
	// redblack : 0 or 1 indicating red or black
	template <typename QType>
	__global__ void kernelGS2d(
		Blob<QType>::wrapper_t q, Blob<QType>::wrapper_const_t g, 
		T omega, uint redblack
	)
	{
		uint y = blockIdx.x * 2u + 1u;
		uint x = threadIdx.x + 1u;

		// Red - all cells with (x + y) % 2 == 0
		y += (x & 1) ^ redblack;

		T v = 0;
		v += q(x - 1u, y);
		v += q(x + 1u, y);
		v += q(x, y - 1u);
		v += q(x, y + 1u);

		q(x, y) = omega * (v - g(x, y)) / (T)(4) + ((T)(1) - omega) * q(x, y);
	}

	// Red-black Gauss-Seidel
	// LAUNCH : block (ny - 2, (nz - 2) / 2), thread (nx - 2)
	// p : nx x ny x nz
	// g : nx x ny x nz
	// omega : SOR coefficient
	// redblack : 0 or 1 indicating red or black
	template <typename QType>
	__global__ void kernelGS3d(
		Blob<QType>::wrapper_t q, Blob<QType>::wrapper_const_t g, 
		T omega, uint redblack
	)
	{
		uint z = blockIdx.y * 2u + 1u;
		uint y = blockIdx.x + 1u;
		uint x = threadIdx.x + 1u;

		// Red - all cells with (x + y + z) % 2 == 0
		z += ((x + y) & 1) ^ redblack;

		T v = 0;
		v += q(x - 1u, y, z);
		v += q(x + 1u, y, z);
		v += q(x, y - 1u, z);
		v += q(x, y + 1u, z);
		v += q(x, y, z - 1u);
		v += q(x, y, z + 1u);

		q(x, y, z) = omega * (v - g(x, y, z)) / (T)(6) 
			+ ((T)(1) - omega) * q(x, y, z);
	}
}

template<typename QType>
void PoissonMethodGS<QType>::operator()(
	Blob<QType>& q, const Blob<QType>& g
	) const
{
	if (q.nz() < 3u)
	{
		for (uint i = 0; i < _iterations; i++)
		{
			kernelGS2d<<<(q.ny() - 2u) / 2u, q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 0
			);
			kernelGS2d<<<(q.ny() - 2u) / 2u, q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 1u
			);
		}
	}
	else
	{
		for (uint i = 0; i < _iterations; i++)
		{
			kernelGS3d<<<dim3(q.ny() - 2u, (q.nz() - 2u) / 2u), q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 0
			);
			kernelGS3d<<<dim3(q.ny() - 2u, (q.nz() - 2u) / 2u), q.nx() - 2u>>>(
				q.wrapper(), g.wrapper_const(), (T)(1), 1u
			);
		}
	}
}

template class PoissonMethodGS<T>;



template<typename QType>
void PoissonMethodVCycle<QType>::operator()(Blob<QType>& q, const Blob<QType>& g) const
{
	if (_buffers.empty() || _buffers.front().shape() != q.shape())
	{
		_buffers.clear();

		for (uint i = 0; i < _levels; i++)
		{
			_buffers.emplace_back(q.shape(), q.gpu_device(), false);
		}
	}
}
