
#include "common.cuh"
#include "pitched_ptr.h"
#include "BlobBase.h"
using namespace ssv;

#include "thrust/copy.h"


void BlobBase::syncGpu2Cpu()
{
	if (_storage_gpu_device < 0) throw error_t::SSV_ERROR_NOT_INITIALIZED;
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);
	if (_data_cpu == nullptr) return;

	copyTo(_data_cpu, storage_t::GPU, storage_t::CPU);
}

void BlobBase::syncCpu2Gpu()
{
	if (_storage_gpu_device < 0) throw error_t::SSV_ERROR_NOT_INITIALIZED;
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);
	if (_data_cpu == nullptr) return;

	copyFrom(_data_cpu, storage_t::CPU, storage_t::GPU);
}

namespace
{
	inline enum hipMemcpyKind make_cudaMemcpyKind(
		BlobBase::storage_t from, BlobBase::storage_t to
	)
	{
		if (from == BlobBase::storage_t::CPU)
		{
			if (to == BlobBase::storage_t::CPU)
				return hipMemcpyHostToHost;
			else
				return hipMemcpyHostToDevice;
		}
		else
		{
			if (to == BlobBase::storage_t::CPU)
				return hipMemcpyDeviceToHost;
			else
				return hipMemcpyDeviceToDevice;
		}
	}
}

void BlobBase::copyTo(void *dst, storage_t from, storage_t to) const
{
	if ((from != storage_t::CPU && from != storage_t::GPU)
		|| (to != storage_t::CPU && to != storage_t::GPU)
		|| dst == nullptr)
	{
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}

	hipPitchedPtr dst_pitched_ptr =
		make_hipPitchedPtr(dst, _nx_in_bytes, _nx_in_bytes, _ny);
	hipPitchedPtr data_cpu_pitched_ptr =
		make_hipPitchedPtr(_data_cpu, _nx_in_bytes, _nx_in_bytes, _ny);
	hipMemcpy3DParms params = { 0 };
	if (from == storage_t::CPU)
		params.srcPtr = data_cpu_pitched_ptr;
	else
		params.srcPtr = _data_gpu;
	params.dstPtr = dst_pitched_ptr;
	params.kind = make_cudaMemcpyKind(from, to);
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		error_t::SSV_ERROR_INVALID_VALUE);
}

void BlobBase::copyTo(hipPitchedPtr *dst, storage_t from, storage_t to) const
{
	if ((from != storage_t::CPU && from != storage_t::GPU)
		|| (to != storage_t::CPU && to != storage_t::GPU)
		|| dst == nullptr || dst->ptr == nullptr)
	{
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}

	hipPitchedPtr data_cpu_pitched_ptr =
		make_hipPitchedPtr(_data_cpu, _nx_in_bytes, _nx_in_bytes, _ny);
	hipMemcpy3DParms params = { 0 };
	if (from == storage_t::CPU)
		params.srcPtr = data_cpu_pitched_ptr;
	else
		params.srcPtr = _data_gpu;
	params.dstPtr = *dst;
	params.kind = make_cudaMemcpyKind(from, to);
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		error_t::SSV_ERROR_INVALID_VALUE);
}

void BlobBase::copyFrom(void *src, storage_t from, storage_t to)
{
	if ((from != storage_t::CPU && from != storage_t::GPU)
		|| src == nullptr)
	{
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}

	hipPitchedPtr src_pitched_ptr =
		make_hipPitchedPtr(src, _nx_in_bytes, _nx_in_bytes, _ny);
	hipPitchedPtr data_cpu_pitched_ptr =
		make_hipPitchedPtr(_data_cpu, _nx_in_bytes, _nx_in_bytes, _ny);
	hipMemcpy3DParms params = { 0 };
	if (to == storage_t::CPU)
		params.dstPtr = data_cpu_pitched_ptr;
	else
		params.dstPtr = _data_gpu;
	params.srcPtr = src_pitched_ptr;
	params.kind = make_cudaMemcpyKind(from, to);
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		error_t::SSV_ERROR_INVALID_VALUE);
}

void BlobBase::copyFrom(hipPitchedPtr *src, storage_t from, storage_t to)
{
	if ((from != storage_t::CPU && from != storage_t::GPU)
		|| src == nullptr || src->ptr == nullptr)
	{
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}

	hipPitchedPtr data_cpu_pitched_ptr =
		make_hipPitchedPtr(_data_cpu, _nx_in_bytes, _nx_in_bytes, _ny);
	hipMemcpy3DParms params = { 0 };
	if (from == storage_t::CPU)
		params.dstPtr = data_cpu_pitched_ptr;
	else
		params.dstPtr = _data_gpu;
	params.srcPtr = *src;
	params.kind = make_cudaMemcpyKind(from, to);
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		error_t::SSV_ERROR_INVALID_VALUE);
}

void BlobBase::destroyTexture(hipTextureObject_t texture_object)
{
	if (_storage_gpu_device < 0) throw error_t::SSV_ERROR_NOT_INITIALIZED;
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);

	if (!texture_object)
	{
		if (_data_texture_default_2d)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_default_2d),
				error_t::SSV_ERROR_UNKNOWN);
			return;
		}
		if (_data_texture_default_3d)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_default_3d),
				error_t::SSV_ERROR_UNKNOWN);
			return;
		}
		for (auto kv : _data_textures)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(kv.second),
				error_t::SSV_ERROR_UNKNOWN);
		}
		if (_data_cuda_array)
		{
			checkCudaErrorAndThrow(hipFreeArray(_data_cuda_array),
				error_t::SSV_ERROR_UNKNOWN);
		}
	}
	else
	{
		if (_data_texture_default_2d == texture_object)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_default_2d),
				error_t::SSV_ERROR_UNKNOWN);
			return;
		}
		if (_data_texture_default_3d == texture_object)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_default_3d),
				error_t::SSV_ERROR_UNKNOWN);
			return;
		}
		for (auto kv : _data_textures)
		{
			if (kv.second == texture_object)
			{
				_data_textures.erase(kv.first);
				checkCudaErrorAndThrow(hipDestroyTextureObject(kv.second),
					error_t::SSV_ERROR_UNKNOWN);
				return;
			}
		}
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}
}

void BlobBase::clear()
{
	if (_data_cpu)
	{
		memset(_data_cpu, 0, size_cpu_in_bytes());
	}
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);
	checkCudaErrorAndThrow(hipMemset(_data_gpu.ptr, 0, size_gpu_in_bytes()),
		error_t::SSV_ERROR_UNKNOWN);
}

hipTextureObject_t BlobBase::_CreateTexture2d(
	const texture_param_t &params
) const
{
	unsigned char dimension;
	struct hipTextureDesc sTexDesc;
	hipChannelFormatDesc sChannelDesc; 
	size_t layer_id;

	std::tie(sTexDesc, sChannelDesc, dimension, layer_id) = params;

	if (dimension != 2u || layer_id >= _nz)
	{
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}

	if (_storage_gpu_device < 0) throw error_t::SSV_ERROR_NOT_INITIALIZED;
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);

	hipResourceDesc sResDesc;
	memset(&sResDesc, 0, sizeof(sResDesc));
	sResDesc.resType = hipResourceTypePitch2D;
	sResDesc.res.pitch2D.desc = sChannelDesc;
	sResDesc.res.pitch2D.devPtr = 
		static_cast<byte *>(_data_gpu.ptr)
		+ layer_id * _data_gpu.pitch * _data_gpu.ysize;
	sResDesc.res.pitch2D.width = _data_gpu.xsize;
	sResDesc.res.pitch2D.height = _data_gpu.ysize;
	sResDesc.res.pitch2D.pitchInBytes = _data_gpu.pitch;

	hipTextureObject_t texture_object = 0;
	checkCudaErrorAndThrow(hipCreateTextureObject(&texture_object, &sResDesc, &sTexDesc, NULL),
		error_t::SSV_ERROR_INVALID_VALUE);

	return texture_object;
}

hipTextureObject_t BlobBase::_CreateTexture3d(
	const texture_param_t &params
) const
{
	unsigned char dimension;
	struct hipTextureDesc sTexDesc;
	hipChannelFormatDesc sChannelDesc;
	size_t layer_id;

	std::tie(sTexDesc, sChannelDesc, dimension, layer_id) = params;

	if (dimension != 3u || layer_id != 0)
	{
		throw error_t::SSV_ERROR_INVALID_VALUE;
	}

	if (_storage_gpu_device < 0) throw error_t::SSV_ERROR_NOT_INITIALIZED;
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);

	if (!_data_cuda_array)
	{
		size_t element_size_in_bytes =
			(sChannelDesc.x + sChannelDesc.y + sChannelDesc.z + sChannelDesc.w) / 8u;
		hipExtent extent_in_elements = make_hipExtent(
			_nx_in_bytes / element_size_in_bytes, _ny, _nz
		);

		checkCudaErrorAndThrow(hipMalloc3DArray(&_data_cuda_array, &sChannelDesc, extent_in_elements),
			error_t::SSV_ERROR_OUT_OF_MEMORY_GPU);
	}

	hipResourceDesc sResDesc;
	memset(&sResDesc, 0, sizeof(sResDesc));
	sResDesc.resType = hipResourceTypeArray;
	sResDesc.res.array.array = _data_cuda_array;

	hipTextureObject_t texture_object = 0;
	checkCudaErrorAndThrow(hipCreateTextureObject(&texture_object, &sResDesc, &sTexDesc, NULL),
		error_t::SSV_ERROR_INVALID_VALUE);

	return texture_object;
}

void BlobBase::_CopyToCudaArray() const
{
	if (_storage_gpu_device < 0) throw error_t::SSV_ERROR_NOT_INITIALIZED;
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);

	hipExtent extent_in_elements;
	hipArrayGetInfo(nullptr, &extent_in_elements, nullptr, _data_cuda_array);

	hipMemcpy3DParms params = { 0 };
	params.srcPtr = _data_gpu;
	params.dstArray = _data_cuda_array;
	params.kind = hipMemcpyDeviceToDevice;
	params.extent = extent_in_elements;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		error_t::SSV_ERROR_INVALID_VALUE);
}

BlobBase::texture_param_t BlobBase::_MakeTextureParam(
	const hipTextureDesc * texDesc, const hipChannelFormatDesc * channelDesc, 
	unsigned char dimension, uint layer_id
)
{
	struct hipTextureDesc sTexDesc;
	if (texDesc == nullptr)
	{
		memset(&sTexDesc, 0, sizeof(sTexDesc));
		sTexDesc.addressMode[0] = hipAddressModeClamp;
		sTexDesc.addressMode[1] = hipAddressModeClamp;
		sTexDesc.addressMode[2] = hipAddressModeClamp;
		sTexDesc.filterMode = hipFilterModeLinear;
		sTexDesc.readMode = hipReadModeElementType;
		sTexDesc.normalizedCoords = 0;
		texDesc = &sTexDesc;
	}

	hipChannelFormatDesc sChannelDesc;
	if (channelDesc == nullptr)
	{
		sChannelDesc = hipCreateChannelDesc<byte>();
		channelDesc = &sChannelDesc;
	}

	return std::make_tuple(*texDesc, *channelDesc, dimension, layer_id);
}

void BlobBase::_InitCuda(int gpu_device)
{
	if (gpu_device < 0)
	{
		_storage_gpu_device = -1;
		memset(&_data_gpu_extent, 0, sizeof(hipExtent));
		memset(&_data_gpu, 0, sizeof(hipPitchedPtr));
		_data_texture_default_2d = 0;
		_data_texture_default_3d = 0;
		_data_textures.clear();
		_data_cuda_array = nullptr;
	}
	else
	{
		_storage_gpu_device = gpu_device;
		_data_gpu_extent = make_hipExtent(_nx_in_bytes,_ny, _nz);

		checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
			error_t::SSV_ERROR_DEVICE_NOT_READY);
		checkCudaErrorAndThrow(hipMalloc3D(&_data_gpu, _data_gpu_extent),
			error_t::SSV_ERROR_OUT_OF_MEMORY_GPU);
		checkCudaErrorAndThrow(hipMemset(_data_gpu.ptr, 0, size_gpu_in_bytes()),
			error_t::SSV_ERROR_UNKNOWN);

		_data_texture_default_2d = 0;
		_data_texture_default_3d = 0;
		_data_textures.clear();
		_data_cuda_array = nullptr;
	}
}

void BlobBase::_CopyCuda(const BlobBase &other, int gpu_device)
{
	_storage_gpu_device = gpu_device;
	_data_gpu_extent = other._data_gpu_extent;

	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		error_t::SSV_ERROR_DEVICE_NOT_READY);
	checkCudaErrorAndThrow(hipMalloc3D(&_data_gpu, _data_gpu_extent),
		error_t::SSV_ERROR_OUT_OF_MEMORY_GPU);

	hipMemcpy3DParms params = { 0 };
	params.srcPtr = other._data_gpu;
	params.dstPtr = _data_gpu;
	params.kind = hipMemcpyDeviceToDevice;
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		error_t::SSV_ERROR_INVALID_VALUE);

	_data_texture_default_2d = 0;
	_data_texture_default_3d = 0;
	_data_textures.clear();
	_data_cuda_array = nullptr;
}

void BlobBase::_MoveCuda(BlobBase &&other)
{
	_storage_gpu_device = other._storage_gpu_device;
	_data_gpu_extent = other._data_gpu_extent;
	_data_gpu = other._data_gpu;
	_data_texture_default_2d = other._data_texture_default_2d;
	_data_texture_default_3d = other._data_texture_default_3d;
	_data_textures = std::move(other._data_textures);
	_data_cuda_array = other._data_cuda_array;

	memset(&other._data_gpu, 0, sizeof(hipPitchedPtr));
	memset(&other._data_gpu_extent, 0, sizeof(hipExtent));
	other._storage_gpu_device = -1;
	other._data_texture_default_2d = 0;
	other._data_texture_default_3d = 0;
	other._data_textures.clear();
	other._data_cuda_array = nullptr;
}

void BlobBase::_DestroyCuda()
{
	if (_storage_gpu_device >= 0)
	{
		checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
			error_t::SSV_ERROR_DEVICE_NOT_READY);
		if (_data_gpu.ptr)
		{
			checkCudaErrorAndThrow(hipFree(_data_gpu.ptr),
				error_t::SSV_ERROR_INVALID_VALUE);
		}
		destroyTexture();
	}

	memset(&_data_gpu, 0, sizeof(hipPitchedPtr));
	memset(&_data_gpu_extent, 0, sizeof(hipExtent));
	_storage_gpu_device = -1;
	_data_texture_default_2d = 0;
	_data_texture_default_3d = 0;
	_data_textures.clear();
	_data_cuda_array = nullptr;
}
