
#include "common.cuh"
#include "BlobBase.h"
using namespace ssv;


void BlobBase::copyToCpu(hipPitchedPtr *from_gpu_data)
{
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		SSV_ERROR_DEVICE_NOT_READY);

	if (from_gpu_data == nullptr)
	{
		from_gpu_data = &_data_gpu;
	}
	hipPitchedPtr data_cpu_pitched_ptr =
		make_hipPitchedPtr(_data_cpu, _nx, _nx, _ny);
	hipMemcpy3DParms params = { 0 };
	params.srcPtr = *from_gpu_data;
	params.dstPtr = data_cpu_pitched_ptr;
	params.kind = hipMemcpyDeviceToHost;
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		SSV_ERROR_INVALID_VALUE);
}

void BlobBase::copyToGpu(void *from_cpu_data)
{
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		SSV_ERROR_DEVICE_NOT_READY);

	if (from_cpu_data == nullptr)
	{
		from_cpu_data = _data_cpu;
	}
	hipPitchedPtr data_cpu_pitched_ptr =
		make_hipPitchedPtr(from_cpu_data, _nx, _nx, _ny);
	hipMemcpy3DParms params = { 0 };
	params.srcPtr = data_cpu_pitched_ptr;
	params.dstPtr = _data_gpu;
	params.kind = hipMemcpyHostToDevice;
	params.extent = _data_gpu_extent;
	checkCudaErrorAndThrow(hipMemcpy3D(&params), 
		SSV_ERROR_INVALID_VALUE);
}

hipTextureObject_t BlobBase::createTexture2d(
	const hipTextureDesc *texDesc,
	const hipChannelFormatDesc *channelDesc,
	size_t layer_id)
{
	if (layer_id >= _nz)
	{
		throw SSV_ERROR_INVALID_VALUE;
	}

	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		SSV_ERROR_DEVICE_NOT_READY);

	if (_data_texture_2d)
	{
		checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_2d),
			SSV_ERROR_UNKNOWN);
		_data_texture_2d = 0;
	}

	struct hipTextureDesc sTexDesc;
	if (texDesc == nullptr)
	{
		memset(&sTexDesc, 0, sizeof(sTexDesc));
		sTexDesc.addressMode[0] = hipAddressModeClamp;
		sTexDesc.addressMode[1] = hipAddressModeClamp;
		sTexDesc.addressMode[2] = hipAddressModeClamp;
		sTexDesc.filterMode = hipFilterModeLinear;
		sTexDesc.readMode = hipReadModeElementType;
		sTexDesc.normalizedCoords = 0;
		texDesc = &sTexDesc;
	}

	hipChannelFormatDesc sChannelDesc;
	if (channelDesc == nullptr)
	{
		sChannelDesc = hipCreateChannelDesc<byte>();
		channelDesc = &sChannelDesc;
	}

	hipResourceDesc sResDesc;
	memset(&sResDesc, 0, sizeof(sResDesc));
	sResDesc.resType = hipResourceTypePitch2D;
	sResDesc.res.pitch2D.desc = *channelDesc;
	sResDesc.res.pitch2D.devPtr = 
		static_cast<byte *>(_data_gpu.ptr)
		+ layer_id * _data_gpu.pitch * _data_gpu.ysize;
	sResDesc.res.pitch2D.width = _data_gpu.xsize;
	sResDesc.res.pitch2D.height = _data_gpu.ysize;
	sResDesc.res.pitch2D.pitchInBytes = _data_gpu.pitch;

	checkCudaErrorAndThrow(hipCreateTextureObject(&_data_texture_2d, &sResDesc, texDesc, NULL),
		SSV_ERROR_INVALID_VALUE);

	return _data_texture_2d;
}

hipTextureObject_t BlobBase::createTexture3d(
	const hipTextureDesc *texDesc,
	const hipChannelFormatDesc *channelDesc
)
{
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		SSV_ERROR_DEVICE_NOT_READY);

	if (_data_texture_3d)
	{
		checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_3d),
			SSV_ERROR_UNKNOWN);
		_data_texture_3d = 0;
	}

	struct hipTextureDesc sTexDesc;
	if (texDesc == nullptr)
	{
		memset(&sTexDesc, 0, sizeof(sTexDesc));
		sTexDesc.addressMode[0] = hipAddressModeClamp;
		sTexDesc.addressMode[1] = hipAddressModeClamp;
		sTexDesc.addressMode[2] = hipAddressModeClamp;
		sTexDesc.filterMode = hipFilterModeLinear;
		sTexDesc.readMode = hipReadModeElementType;
		sTexDesc.normalizedCoords = 0;
		texDesc = &sTexDesc;
	}

	hipChannelFormatDesc sChannelDesc;
	if (channelDesc == nullptr)
	{
		sChannelDesc = hipCreateChannelDesc<byte>();
		channelDesc = &sChannelDesc;
	}

	size_t element_size_in_bytes = 
		(channelDesc->x + channelDesc->y + channelDesc->z + channelDesc->w) / 8u;
	hipExtent extent_in_elements = make_hipExtent(
		_nx / element_size_in_bytes, _ny, _nz
	);

	checkCudaErrorAndThrow(hipMalloc3DArray(&_data_cuda_array, channelDesc, extent_in_elements),
		SSV_ERROR_OUT_OF_MEMORY_GPU);

	hipResourceDesc sResDesc;
	memset(&sResDesc, 0, sizeof(sResDesc));
	sResDesc.resType = hipResourceTypeArray;
	sResDesc.res.array.array = _data_cuda_array;

	checkCudaErrorAndThrow(hipCreateTextureObject(&_data_texture_3d, &sResDesc, &sTexDesc, NULL),
		SSV_ERROR_INVALID_VALUE);

	return data_texture_3d();
}

hipTextureObject_t BlobBase::data_texture_3d() const
{
	checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
		SSV_ERROR_DEVICE_NOT_READY);

	hipExtent extent_in_elements;
	hipArrayGetInfo(nullptr, &extent_in_elements, nullptr, _data_cuda_array);

	hipMemcpy3DParms params = { 0 };
	params.srcPtr = _data_gpu;
	params.dstArray = _data_cuda_array;
	params.kind = hipMemcpyDeviceToDevice;
	params.extent = extent_in_elements;
	checkCudaErrorAndThrow(hipMemcpy3D(&params),
		SSV_ERROR_INVALID_VALUE);

	return _data_texture_3d;
}

void BlobBase::_InitCuda(int gpu_device)
{
	_storage_gpu_device = gpu_device;
	_data_gpu_extent.width = _nx;
	_data_gpu_extent.height = _ny;
	_data_gpu_extent.depth = _nz;

	if (gpu_device >= 0)
	{
		checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
			SSV_ERROR_DEVICE_NOT_READY);
		checkCudaErrorAndThrow(hipMalloc3D(&_data_gpu, _data_gpu_extent),
			SSV_ERROR_OUT_OF_MEMORY_GPU);
	}
}

void BlobBase::_DestroyCuda()
{
	if (_storage_gpu_device >= 0)
	{
		checkCudaErrorAndThrow(hipSetDevice(_storage_gpu_device),
			SSV_ERROR_DEVICE_NOT_READY);
		if (_data_gpu.ptr)
		{
			checkCudaErrorAndThrow(hipFree(_data_gpu.ptr),
				SSV_ERROR_INVALID_VALUE);
		}
		if (_data_texture_2d)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_2d),
				SSV_ERROR_UNKNOWN);
		}
		if (_data_texture_3d)
		{
			checkCudaErrorAndThrow(hipDestroyTextureObject(_data_texture_3d),
				SSV_ERROR_UNKNOWN);
		}
	}

	memset(&_data_gpu, 0, sizeof(hipPitchedPtr));
	memset(&_data_gpu_extent, 0, sizeof(hipExtent));
	_storage_gpu_device = -1;
	_data_texture_2d = 0;
	_data_texture_3d = 0;
	_data_cuda_array = nullptr;
}
