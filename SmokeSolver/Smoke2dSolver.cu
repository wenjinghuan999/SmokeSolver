#include "hip/hip_runtime.h"

#include "common.cuh"
#include "Smoke2dSolver.h"
using namespace ssv;

#include "pitched_ptr.h"

#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
using thrust::placeholders::_1;

void ssv::Smoke2dSolver::_InitCuda()
{
	hipSetDevice(0);

	_data.setSize(_nx, _ny);
	_data.setSize(5, 2, 3);
}


static __global__ void kernelWWW(
	hipTextureObject_t texObj, hipPitchedPtr anotherptr
	)
{
	size_t j = blockIdx.x;
	size_t i = threadIdx.x;

	float u = i / (float)1; 
	float v = j / (float)1;

	float *p = (float *)anotherptr.ptr;
	p[j * anotherptr.pitch / sizeof(T) + i] = tex3D<float>(texObj, u, v, 2);
}


void Smoke2dSolver::_StepCuda()
{
	hipSetDevice(0);

	T *p = _data.data_cpu();
	for (size_t k = 0; k < 3; k++)
	{
		for (size_t j = 0; j < _ny; j++)
		{
			for (size_t i = 0; i < _nx; i++)
			{
				p[k * _ny * _nx + j * _nx + i] = i * 100 + j * 10 + k;
			}
		}
	}

	for (size_t k = 0; k < 3; k++)
	{
		for (size_t j = 0; j < _ny; j++)
		{
			for (size_t i = 0; i < _nx; i++)
			{
				std::cout << p[k * _ny * _nx + j * _nx + i]  << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}
	for (size_t k = 0; k < 3 * _nx * _ny; k++)
	{
		std::cout << p[k] << " ";
	}
	std::cout << std::endl;
	_data.copyToGpu();

	hipPitchedPtr *ppp = _data.data_gpu_cuda_pitched_ptr();
	T *pd = _data.data_gpu_raw();
	Print(pd, _data.pitch_in_elements() * _data.ny() * _data.nz(), "data:\n");

	Blob<T> another;
	another.setSize(5, 2, 3);
	hipPitchedPtr *ppap = another.data_gpu_cuda_pitched_ptr();

	hipTextureObject_t texObj = _data.createTexture3d();
	
	kernelWWW<<<2, 5>>>(texObj, *ppap);

	hipDeviceSynchronize();

	Print(another.data_gpu_raw(), ppap->pitch / sizeof(T) * ppap->ysize * 3, "data:\n");
	another.copyToCpu();
	p = another.data_cpu();

	std::cout << "another" << std::endl;
	for (size_t k = 0; k < 3; k++)
	{
		for (size_t j = 0; j < _ny; j++)
		{
			for (size_t i = 0; i < _nx; i++)
			{
				std::cout << p[k * _ny * _nx + j * _nx + i] << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}


	thrust::transform(_data.data_gpu(), _data.data_gpu() + _nx*_ny*3, _data.data_gpu(), 1.f + _1 * _1);
	Print(pd, _data.pitch_in_elements() * _data.ny() * _data.nz(), "data:\n");
	_data.copyToCpu();
	p = _data.data_cpu();





	std::cout << std::endl;
	for (size_t k = 0; k < 3; k++)
	{
		for (size_t j = 0; j < _ny; j++)
		{
			for (size_t i = 0; i < _nx; i++)
			{
				std::cout << p[k * _ny * _nx + j * _nx + i] << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}

	for (size_t k = 0; k < _nx * _ny * 3; k++)
	{
		std::cout << p[k] << " ";
	}
	std::cout << std::endl;

	//texObj = _data.data_texture_3d();
	kernelWWW<<<2, 5>>>(texObj, *ppap);

	hipDeviceSynchronize();

	Print(another.data_gpu_raw(), another.pitch_in_elements() * another.ny() * another.nz(), "data:\n");
	another.copyToCpu();
	p = another.data_cpu();

	std::cout << "another" << std::endl;
	for (size_t k = 0; k < 3; k++)
	{
		for (size_t j = 0; j < _ny; j++)
		{
			for (size_t i = 0; i < _nx; i++)
			{
				std::cout << p[k * _ny * _nx + j * _nx + i] << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}

	another.reset();
	another.setSize(5, 2, 3);

	texObj = _data.data_texture_3d();
	kernelWWW << <2, 5 >> >(texObj, *ppap);

	hipDeviceSynchronize();

	Print(another.data_gpu_raw(), ppap->pitch / sizeof(T) * ppap->ysize * 3, "data:\n");
	another.copyToCpu();
	p = another.data_cpu();

	std::cout << "another" << std::endl;
	for (size_t k = 0; k < 3; k++)
	{
		for (size_t j = 0; j < _ny; j++)
		{
			for (size_t i = 0; i < _nx; i++)
			{
				std::cout << p[k * _ny * _nx + j * _nx + i] << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}

}
